
#include <hip/hip_runtime.h>
// Tests CUDA compilation pipeline construction in Driver.
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// Simple compilation case. Compile device-side to PTX assembly and make sure
// we use it on the host side.
// RUN: %clang -### -target x86_64-linux-gnu -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix NOLINK %s

// Typical compilation + link case.
// RUN: %clang -### -target x86_64-linux-gnu %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix LINK %s

// Verify that --cuda-host-only disables device-side compilation, but doesn't
// disable host-side compilation/linking.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// Verify that --cuda-device-only disables host-side compilation and linking.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// Check that the last of --cuda-compile-host-device, --cuda-host-only, and
// --cuda-device-only wins.

// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:    --cuda-host-only %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-compile-host-device \
// RUN:    --cuda-host-only %s 2>&1 \
// RUN: | FileCheck -check-prefix NODEVICE -check-prefix HOST \
// RUN:    -check-prefix NOINCLUDES-DEVICE -check-prefix LINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only \
// RUN:    --cuda-device-only %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-compile-host-device \
// RUN:    --cuda-device-only %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix NOHOST -check-prefix NOLINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-host-only \
// RUN:   --cuda-compile-host-device %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix LINK %s

// RUN: %clang -### -target x86_64-linux-gnu --cuda-device-only \
// RUN:   --cuda-compile-host-device %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix LINK %s

// Verify that --cuda-gpu-arch option passes the correct GPU archtecture to
// device compilation.
// RUN: %clang -### -target x86_64-linux-gnu --cuda-gpu-arch=sm_35 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix DEVICE-SM35 -check-prefix HOST \
// RUN:    -check-prefix INCLUDES-DEVICE -check-prefix NOLINK %s

// Verify that there is one device-side compilation per --cuda-gpu-arch args
// and that all results are included on the host side.
// RUN: %clang -### -target x86_64-linux-gnu \
// RUN:   --cuda-gpu-arch=sm_35 --cuda-gpu-arch=sm_30 -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix DEVICE2 -check-prefix DEVICE-SM35 \
// RUN:    -check-prefix DEVICE2-SM30 -check-prefix HOST \
// RUN:    -check-prefix HOST-NOSAVE -check-prefix INCLUDES-DEVICE \
// RUN:    -check-prefix NOLINK %s

// Verify that device-side results are passed to the correct tool when
// -save-temps is used.
// RUN: %clang -### -target x86_64-linux-gnu -save-temps -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-SAVE \
// RUN:    -check-prefix HOST -check-prefix HOST-SAVE -check-prefix NOLINK %s

// Verify that device-side results are passed to the correct tool when
// -fno-integrated-as is used.
// RUN: %clang -### -target x86_64-linux-gnu -fno-integrated-as -c %s 2>&1 \
// RUN: | FileCheck -check-prefix DEVICE -check-prefix DEVICE-NOSAVE \
// RUN:    -check-prefix HOST -check-prefix HOST-NOSAVE \
// RUN:    -check-prefix HOST-AS -check-prefix NOLINK %s

// Match device-side preprocessor and compiler phases with -save-temps.
// DEVICE-SAVE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-SAVE-SAME: "-aux-triple" "x86_64--linux-gnu"
// DEVICE-SAVE-SAME: "-fcuda-is-device"
// DEVICE-SAVE-SAME: "-x" "cuda"

// DEVICE-SAVE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-SAVE-SAME: "-aux-triple" "x86_64--linux-gnu"
// DEVICE-SAVE-SAME: "-fcuda-is-device"
// DEVICE-SAVE-SAME: "-x" "cuda-cpp-output"

// Match the job that produces PTX assembly.
// DEVICE: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE-NOSAVE-SAME: "-aux-triple" "x86_64--linux-gnu"
// DEVICE-SAME: "-fcuda-is-device"
// DEVICE-SM35-SAME: "-target-cpu" "sm_35"
// DEVICE-SAME: "-o" "[[PTXFILE:[^"]*]]"
// DEVICE-NOSAVE-SAME: "-x" "cuda"
// DEVICE-SAVE-SAME: "-x" "ir"

// Match the call to ptxas (which assembles PTX to SASS).
// DEVICE:ptxas
// DEVICE-SM35-DAG: "--gpu-name" "sm_35"
// DEVICE-DAG: "--output-file" "[[CUBINFILE:[^"]*]]"
// DEVICE-DAG: "[[PTXFILE]]"

// Match another device-side compilation.
// DEVICE2: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// DEVICE2-SAME: "-aux-triple" "x86_64--linux-gnu"
// DEVICE2-SAME: "-fcuda-is-device"
// DEVICE2-SM30-SAME: "-target-cpu" "sm_30"
// DEVICE2-SAME: "-o" "[[GPUBINARY2:[^"]*]]"
// DEVICE2-SAME: "-x" "cuda"

// Match no device-side compilation.
// NODEVICE-NOT: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// NODEVICE-NOT: "-fcuda-is-device"

// INCLUDES-DEVICE:fatbinary
// INCLUDES-DEVICE-DAG: "--create" "[[FATBINARY:[^"]*]]"
// INCLUDES-DEVICE-DAG: "--image=profile=sm_{{[0-9]+}},file=[[CUBINFILE]]"
// INCLUDES-DEVICE-DAG: "--image=profile=compute_{{[0-9]+}},file=[[PTXFILE]]"

// Match host-side preprocessor job with -save-temps.
// HOST-SAVE: "-cc1" "-triple" "x86_64--linux-gnu"
// HOST-SAVE-SAME: "-aux-triple" "nvptx64-nvidia-cuda"
// HOST-SAVE-NOT: "-fcuda-is-device"
// HOST-SAVE-SAME: "-x" "cuda"

// Match host-side compilation.
// HOST: "-cc1" "-triple" "x86_64--linux-gnu"
// HOST-SAME: "-aux-triple" "nvptx64-nvidia-cuda"
// HOST-NOT: "-fcuda-is-device"
// HOST-SAME: "-o" "[[HOSTOUTPUT:[^"]*]]"
// HOST-NOSAVE-SAME: "-x" "cuda"
// HOST-SAVE-SAME: "-x" "cuda-cpp-output"
// INCLUDES-DEVICE-SAME: "-fcuda-include-gpubinary" "[[FATBINARY]]"

// Match external assembler that uses compilation output.
// HOST-AS: "-o" "{{.*}}.o" "[[HOSTOUTPUT]]"

// Match no GPU code inclusion.
// NOINCLUDES-DEVICE-NOT: "-fcuda-include-gpubinary"

// Match no host compilation.
// NOHOST-NOT: "-cc1" "-triple"
// NOHOST-NOT: "-x" "cuda"

// Match linker.
// LINK: "{{.*}}{{ld|link}}{{(.exe)?}}"
// LINK-SAME: "[[HOSTOUTPUT]]"

// Match no linker.
// NOLINK-NOT: "{{.*}}{{ld|link}}{{(.exe)?}}"
